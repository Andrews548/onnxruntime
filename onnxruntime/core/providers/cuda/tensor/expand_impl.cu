#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "expand_impl.h"
#include "core/providers/cuda/cuda_common.h"

namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void ExpandKernel2D(
    const int N,
    const T* input_data,
    T* output_data,
    const fast_divmod fdm_output_stride0, 
    const int input_view_stride0,
    const int input_view_stride1) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);

  int dim0, dim1;
  fdm_output_stride0.divmod(id, dim0, dim1);
  output_data[id] = input_data[dim0 * input_view_stride0 + dim1 * input_view_stride1];
}

template <typename T>
__global__ void ExpandKernel(
  const size_t rank,
  const int N,
  const void* input_data,
  void* output_data,
  const fast_divmod* fdm_output_strides,
  const int64_t* input_view_strides) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);

  int dim, r = id, input_index = 0;
  for (int i = 0; i < rank; ++i) {
    fdm_output_strides[i].divmod(r, dim, r);
    input_index += dim * input_view_strides[i];
  }
  output_data[id] = input_data[input_index];
}

Status ExpandByFill(const size_t element_size, const int N, const void* input_data, void* output_data)
{
#define FILLON(TYPE)                                                                                          \
  case sizeof(TYPE):                                                                                        \
    cuda::Fill(reinterpret_cast<TYPE*>(output_data), *(reinterpret_cast<const TYPE*>(input_data)), N);  \
    break

  switch (element_size) {
    FILLON(int8_t);
    FILLON(int16_t);
    FILLON(int32_t);
    FILLON(int64_t);
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for Expand operator");
  }
  return Status::OK();
}

Status Expand2D(
  const size_t element_size,    
  const int N,
  const void* input_data,
  void* output_data,
  const fast_divmod fdm_output_stride0, 
  const int input_view_stride0,
  const int input_view_stride1) {
#define EXPAND2D_ON(TYPE)                                                                                          \
    case sizeof(TYPE):                                                                                        \
      ExpandKernel2D<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>( \
        N, *(reinterpret_cast<const TYPE*>(input_data)), reinterpret_cast<TYPE*>(output_data), \
	fdm_output_stride0, input_view_stride0, input_view_stride1);  \
      break

  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  switch (element_size) {
    EXPAND2D_ON(int8_t);
    EXPAND2D_ON(int16_t);
    EXPAND2D_ON(int32_t);
    EXPAND2D_ON(int64_t);
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for Expand operator");
  }
  return Status::OK();
}

Status ExpandImpl(
  const size_t element_size,
  const int N_output,
  const int N_input,
  const void* input_data,
  void* output_data,
  CudaKernel::CudaAsyncBuffer<fast_divmod>& fdm_output_strides, 
  CudaKernel::CudaAsyncBuffer<int64_t>& input_view_strides)
{
  const int rank = static_cast<int>(fdm_output_strides.Count());
  if (rank == 1) {
    if (N_input == N_output) {
      CUDA_RETURN_IF_ERROR(hipMemcpyAsync(output_data, input_data, N * element_size, hipMemcpyDeviceToDevice, 0));
    }
    else { // N_input == 1
      return ExpandByFill(element_size, N_output, input_data, output_data);
    }
  }
  else if (rank == 2) {
    return Expand2D(element_size, N_output, input_data, output_data,
      fdm_output_strides.CpuSpan()[0], 
      input_view_strides.CpuSpan[0], input_view_strides.CpuSpan()[1]);
  }

  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  fdm_output_strides.CopyToGpu();
  input_view_strides.CopyToGpu();
  #define EXPAND_ON(TYPE)                                                                                          \
    case sizeof(TYPE):                                                                                        \
      ExpandKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>( \
        rank, N_output, *(reinterpret_cast<const TYPE*>(input_data)), reinterpret_cast<TYPE*>(output_data), \
        fdm_output_strides.GpuPtr(), input_view_strides.GpuPtr());  \
      break

  switch (element_size) {
    EXPAND_ON(uint8_t);
    EXPAND_ON(uint16_t);
    EXPAND_ON(uint32_t);
    EXPAND_ON(uint64_t);
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for Expand operator");
  }
  return Status::OK();
}


}  // namespace cuda
}  // namespace onnxruntime
